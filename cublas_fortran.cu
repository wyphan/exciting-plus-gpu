#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsolver.h"

extern "C" int f_cublasCreate(hipblasHandle_t **handle)
{
    *handle = (hipblasHandle_t*)malloc(sizeof(hipblasHandle_t));
    return hipblasCreate(*handle);
}

extern "C" int f_cublasZgemm(hipblasHandle_t *handle,
               hipblasOperation_t transa, hipblasOperation_t transb, 
              int m, int n, int k, 
              const hipDoubleComplex *alpha,
              const hipDoubleComplex *A, int lda, 
              const hipDoubleComplex *B, int ldb,
              const hipDoubleComplex *beta, 
              hipDoubleComplex *C, int ldc)
{
    return hipblasZgemm(*handle,transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc);
}

extern "C" int f_cublasZgemmBatched(hipblasHandle_t *handle,
               hipblasOperation_t transa, hipblasOperation_t transb,
              int m, int n, int k,
              const hipDoubleComplex *alpha,
              const hipDoubleComplex **A, int lda,
              const hipDoubleComplex **B, int ldb,
              const hipDoubleComplex *beta,
              hipDoubleComplex **C, int ldc,
              int batch_count)
{
    return hipblasZgemmBatched(*handle,transa,transb,m,n,k,alpha,A,lda,B,ldb,beta,C,ldc,batch_count);
}

extern "C" void f_cublasDestroy(hipblasHandle_t *handle)
{
    hipblasDestroy(*handle);
    free(handle);
}

extern "C" int f_cudaStreamCreate(hipStream_t **stream)
{
    *stream = (hipStream_t *) malloc(sizeof(hipStream_t));
    return hipStreamCreate(*stream);
}

extern "C" int f_cublasSetStream(hipblasHandle_t *handle, hipStream_t *streamid)
{
    return hipblasSetStream(*handle, *streamid);
}

extern "C" void f_cudaStreamDestroy(hipStream_t *stream)
{
    hipStreamDestroy(*stream);
}

extern "C" void f_addOffsetToPtr(size_t* array, size_t offset)
{
    array[0] = array[0] + offset;
}

extern "C" void f_printValue(void** address)
{
    //printf(" Write something out!!! PLEASE!\n\n\n");
    printf("     value is %zu\n", ( (size_t*)(*address) ) );
}

//-----------------------------------------------------------------------------

extern "C" int f_cusolverDnCreate(hipsolverHandle_t **handle)
{
    *handle = (hipsolverHandle_t*)malloc(sizeof(hipsolverHandle_t));
    return hipsolverDnCreate(*handle);
}

extern "C" void f_cusolverDnDestroy(hipsolverHandle_t *handle)
{
    hipsolverDnDestroy(*handle);
    free(handle);
}

extern "C" int f_cusolverDnSetStream(hipsolverHandle_t *handle,
				     hipStream_t *streamId)
{
    return hipsolverSetStream(*handle, *streamId);
}

extern "C" int f_cusolverDnZgetrf_bufferSize(hipsolverHandle_t *handle,
					     int m, int n,
					     hipDoubleComplex *A, int lda,
					     int *Lwork)
{
  return hipsolverDnZgetrf_bufferSize(*handle, m, n, A, lda, Lwork);
}

extern "C" int f_cusolverDnZgetrf(hipsolverHandle_t *handle,
				  int m, int n,
				  hipDoubleComplex *A, int lda,
				  hipDoubleComplex *Workspace,
				  int *devIpiv,
				  int *devInfo)
{
  return hipsolverDnZgetrf(*handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

extern "C" int f_cublasZtrsm(hipblasHandle_t *handle,
			     hipblasSideMode_t side, hipblasFillMode_t uplo,
			     hipblasOperation_t transa, hipblasDiagType_t diag,
			     int m, int n,
			     const hipDoubleComplex *alpha,
			     const hipDoubleComplex *A, int lda, 
			     hipDoubleComplex *B, int ldb)
{
  return hipblasZtrsm(*handle, side, uplo, transa, diag, m, n, alpha, A, lda,
		     B, ldb);
}
